#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

//
// This sample demonstrates the use of streams for concurrent execution. It also illustrates how to 
// introduce dependencies between CUDA streams with the new hipStreamWaitEvent function introduced 
// in CUDA 3.2.
//
// Devices of compute capability 1.x will run the kernels one after another
// Devices of compute capability 2.0 or higher can overlap the kernels
//
#include <stdio.h>
//#include <cutil_inline.h>
#include <sdkHelper.h>  // helper for shared functions common to CUDA SDK samples
#include <shrUtils.h>
#include <shrQATest.h>

const char *sSDKsample = "concurrentKernels";

// This is a kernel that does no real work but runs at least for a specified number of clocks
__global__ void clock_block(clock_t* d_o, clock_t clock_count)
{ 
	clock_t start_clock = clock();
	
	clock_t clock_offset = 0;

	while( clock_offset < clock_count ) {
		clock_offset = clock() - start_clock;
	}

	d_o[0] = clock_offset;
}


// Single warp reduction kernel
__global__ void sum(clock_t* d_clocks, int N)
{
	__shared__ clock_t s_clocks[32];

	clock_t my_sum = 0;

	for( int i = threadIdx.x; i < N; i+= blockDim.x ) {
		my_sum += d_clocks[i];
	}

	s_clocks[threadIdx.x] = my_sum;
	syncthreads();	

	for( int i=16; i>0; i/=2) {
		if( threadIdx.x < i ) {
			s_clocks[threadIdx.x] += s_clocks[threadIdx.x + i];
		}
		syncthreads();	
	}	

	d_clocks[0] = s_clocks[0];
}


////////////////////////////////////////////////////////////////////////////////
// These are CUDA Helper functions

// This will output the proper CUDA error strings in the event that a CUDA host call returns an error
#define checkCudaErrors(err)  __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line )
{
	if(hipSuccess != err)
	{
		fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",file, line, (int)err, hipGetErrorString( err ) );
		exit(-1);        
	}
}

// This will output the proper error string when calling hipGetLastError
#define getLastCudaError(msg)      __getLastCudaError (msg, __FILE__, __LINE__)

inline void __getLastCudaError(const char *errorMessage, const char *file, const int line )
{
	hipError_t err = hipGetLastError();
	if (hipSuccess != err)
	{
		fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",
			file, line, errorMessage, (int)err, hipGetErrorString( err ) );
		exit(-1);
	}
}

// General GPU Device CUDA Initialization
int gpuDeviceInit(int devID)
{
	int deviceCount;
	checkCudaErrors(hipGetDeviceCount(&deviceCount));

	if (deviceCount == 0)
	{
		fprintf(stderr, "gpuDeviceInit() CUDA error: no devices supporting CUDA.\n");
		exit(-1);
	}

	if (devID < 0)
		devID = 0;

	if (devID > deviceCount-1)
	{
		fprintf(stderr, "\n");
		fprintf(stderr, ">> %d CUDA capable GPU device(s) detected. <<\n", deviceCount);
		fprintf(stderr, ">> gpuDeviceInit (-device=%d) is not a valid GPU device. <<\n", devID);
		fprintf(stderr, "\n");
		return -devID;
	}

	hipDeviceProp_t deviceProp;
	checkCudaErrors( hipGetDeviceProperties(&deviceProp, devID) );

	if (deviceProp.major < 1)
	{
		fprintf(stderr, "gpuDeviceInit(): GPU device does not support CUDA.\n");
		exit(-1);                                                  
	}

	checkCudaErrors( hipSetDevice(devID) );
	printf("gpuDeviceInit() CUDA Device [%d]: \"%s\n", devID, deviceProp.name);

	return devID;
}

// This function returns the best GPU (with maximum GFLOPS)
int gpuGetMaxGflopsDeviceId()
{
	int current_device     = 0, sm_per_multiproc  = 0;
	int max_compute_perf   = 0, max_perf_device   = 0;
	int device_count       = 0, best_SM_arch      = 0;
	hipDeviceProp_t deviceProp;
	hipGetDeviceCount( &device_count );

	// Find the best major SM Architecture GPU device
	while (current_device < device_count)
	{
		hipGetDeviceProperties( &deviceProp, current_device );
		if (deviceProp.major > 0 && deviceProp.major < 9999)
		{
			best_SM_arch = MAX(best_SM_arch, deviceProp.major);
		}
		current_device++;
	}

	// Find the best CUDA capable GPU device
	current_device = 0;
	while( current_device < device_count )
	{
		hipGetDeviceProperties( &deviceProp, current_device );
		if (deviceProp.major == 9999 && deviceProp.minor == 9999)
		{
			sm_per_multiproc = 1;
		}
		else
		{
			sm_per_multiproc = _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor);
		}

		int compute_perf  = deviceProp.multiProcessorCount * sm_per_multiproc * deviceProp.clockRate;

		if( compute_perf  > max_compute_perf )
		{
			// If we find GPU with SM major > 2, search only these
			if ( best_SM_arch > 2 )
			{
				// If our device==dest_SM_arch, choose this, or else pass
				if (deviceProp.major == best_SM_arch)
				{
					max_compute_perf  = compute_perf;
					max_perf_device   = current_device;
				}
			}
			else
			{
				max_compute_perf  = compute_perf;
				max_perf_device   = current_device;
			}
		}
		++current_device;
	}
	return max_perf_device;
}

// Initialization code to find the best CUDA Device
int findCudaDevice(int argc, const char **argv)
{
	hipDeviceProp_t deviceProp;
	int devID = 0;
	// If the command-line has a device number specified, use it
	if (checkCmdLineFlag(argc, argv, "device"))
	{
		devID = getCmdLineArgumentInt(argc, argv, "device=");
		if (devID < 0)
		{
			printf("Invalid command line parameter\n ");
			exit(-1);
		}
		else
		{
			devID = gpuDeviceInit(devID);
			if (devID < 0)
			{
				printf("exiting...\n");
				shrQAFinishExit(argc, (const char **)argv, QA_FAILED);
				exit(-1);
			}
		}
	}
	else
	{
		// Otherwise pick the device with highest Gflops/s
		devID = gpuGetMaxGflopsDeviceId();
		checkCudaErrors( hipSetDevice( devID ) );
		checkCudaErrors( hipGetDeviceProperties(&deviceProp, devID) );
		printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);
	}
	return devID;
}
// end of CUDA Helper Functions

int main(int argc, char **argv)
{
    int nkernels = 8;               // number of concurrent kernels
    int nstreams = nkernels + 1;    // use one more stream than concurrent kernel
    int nbytes = nkernels * sizeof(clock_t);   // number of data bytes
    float kernel_time = 10; // time the kernel should run in ms
    float elapsed_time;   // timing variables
    int cuda_device = 0;

    shrQAStart(argc, argv); 

    // get number of kernels if overridden on the command line
    if (checkCmdLineFlag(argc, (const char **)argv, "nkernels")) {
        nkernels = getCmdLineArgumentInt(argc, (const char **)argv, "nkernels");
        nstreams = nkernels + 1;
    }

    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    cuda_device = findCudaDevice(argc, (const char**)argv);

    hipDeviceProp_t deviceProp;
    checkCudaErrors( hipGetDevice(&cuda_device));	

    checkCudaErrors( hipGetDeviceProperties(&deviceProp, cuda_device) );
    if( (deviceProp.concurrentKernels == 0 )) {
        shrLog("> GPU does not support concurrent kernel execution\n");
        shrLog("  CUDA kernel runs will be serialized\n");
    }

    shrLog("> Detected Compute SM %d.%d hardware with %d multi-processors\n", 
           deviceProp.major, deviceProp.minor, deviceProp.multiProcessorCount); 

    // allocate host memory
    clock_t *a = 0;                     // pointer to the array data in host memory
    checkCudaErrors( hipHostMalloc((void**)&a, nbytes) ); 

    // allocate device memory
    clock_t *d_a = 0;             // pointers to data and init value in the device memory
    checkCudaErrors( hipMalloc((void**)&d_a, nbytes) );

    // allocate and initialize an array of stream handles
    hipStream_t *streams = (hipStream_t*) malloc(nstreams * sizeof(hipStream_t));
    for(int i = 0; i < nstreams; i++)
        checkCudaErrors( hipStreamCreate(&(streams[i])) );

    // create CUDA event handles
    hipEvent_t start_event, stop_event;
    checkCudaErrors( hipEventCreate(&start_event) );
    checkCudaErrors( hipEventCreate(&stop_event) );

   
    // the events are used for synchronization only and hence do not need to record timings
    // this also makes events not introduce global sync points when recorded which is critical to get overlap 
    hipEvent_t *kernelEvent;
    kernelEvent = (hipEvent_t*) malloc(nkernels * sizeof(hipEvent_t));
    for(int i = 0; i < nkernels; i++)
        checkCudaErrors( hipEventCreateWithFlags(&(kernelEvent[i]), hipEventDisableTiming) );

    //////////////////////////////////////////////////////////////////////
    // time execution with nkernels streams
    clock_t total_clocks = 0;
    clock_t time_clocks = kernel_time * deviceProp.clockRate;
	
    hipEventRecord(start_event, 0);
    // queue nkernels in separate streams and record when they are done
    for( int i=0; i<nkernels; ++i)
    {
        clock_block<<<1,1,0,streams[i]>>>(&d_a[i], time_clocks );
        total_clocks += time_clocks;
        checkCudaErrors( hipEventRecord(kernelEvent[i], streams[i]) );
	
        // make the last stream wait for the kernel event to be recorded
        checkCudaErrors( hipStreamWaitEvent(streams[nstreams-1], kernelEvent[i],0) );
    }

    // queue a sum kernel and a copy back to host in the last stream. 
    // the commands in this stream get dispatched as soon as all the kernel events have been recorded
    sum<<<1,32,0,streams[nstreams-1]>>>(d_a, nkernels);
    checkCudaErrors( hipMemcpyAsync(a, d_a, sizeof(clock_t), hipMemcpyDeviceToHost, streams[nstreams-1]) );
 
    // at this point the CPU has dispatched all work for the GPU and can continue processing other tasks in parallel

    // in this sample we just wait until the GPU is done
    checkCudaErrors( hipEventRecord(stop_event, 0) );
    checkCudaErrors( hipEventSynchronize(stop_event) );
    checkCudaErrors( hipEventElapsedTime(&elapsed_time, start_event, stop_event) );
    
    shrLog("Expected time for serial execution of %d kernels = %.3fs\n", nkernels, nkernels * kernel_time/1000.0f);
    shrLog("Expected time for concurrent execution of %d kernels = %.3fs\n", nkernels, kernel_time/1000.0f);
    shrLog("Measured time for sample = %.3fs\n", elapsed_time/1000.0f);

    bool bTestResult  = (a[0] > total_clocks);

    // release resources
    for(int i = 0; i < nkernels; i++) {
        hipStreamDestroy(streams[i]); 
        hipEventDestroy(kernelEvent[i]);
    }
    free(streams);
    free(kernelEvent);

    hipEventDestroy(start_event);
    hipEventDestroy(stop_event);
    hipHostFree(a);
    hipFree(d_a);

    hipDeviceReset();
    shrQAFinishExit(argc, (const char **)argv, (bTestResult) ? QA_PASSED : QA_FAILED);
}
