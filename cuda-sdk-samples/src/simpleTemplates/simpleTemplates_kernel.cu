#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/* This sample is a templatized version of the template project.
 * It also shows how to correctly templatize dynamically allocated shared
 * memory arrays.
 * Device code.
 */

#ifndef _TEMPLATE_KERNEL_H_
#define _TEMPLATE_KERNEL_H_

#include <stdio.h>
#include "sharedmem.cuh"

#define		IPL				4// 1 2 4 8

////////////////////////////////////////////////////////////////////////////////
//! Simple test kernel for device functionality
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
template<class T>
__global__ void
testKernel( T* g_idata, T* g_odata) 
{
	int block = blockIdx.x + blockIdx.y * gridDim.x;
	int index = threadIdx.x + IPL*block * blockDim.x;

	T a[IPL];

	for(int i=0; i<IPL; i++)
		a[i] = g_idata[index+i * blockDim.x];

	for(int i=0; i<IPL; i++)
		g_odata[index+i * blockDim.x] = a[i];
	
}

#endif // #ifndef _TEMPLATE_KERNEL_H_
